#include "hip/hip_runtime.h"
#include "lanczos_resample.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <>
#include <cmath>

using namespace std;

__device__ double lanczos_kernel(double x, int a) {
    if (x == 0.0) return 1.0;
    if (x <= -a || x >= a) return 0.0;
    double pix = PI * x;
    return (sin(pix) * sin(pix / a)) / (pix * pix / a);
}



// CUDA kernel for Lanczos resampling
__global__ void lanczos_resample_kernel(
    const unsigned char* input,
    unsigned char* output,
    int in_width,
    int in_height,
    int out_width,
    int out_height,
    int channels,
    int a  // Include 'a' parameter
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;  // Output pixel x
    int y = blockIdx.y * blockDim.y + threadIdx.y;  // Output pixel y

    if (x >= out_width || y >= out_height)
        return;

    int idx_out = (y * out_width + x) * channels;

    double sum[3] = { 0.0, 0.0, 0.0 };
    double sum_weight = 0.0;

    // Compute the corresponding input coordinate
    double scale_x = static_cast<double>(out_width) / in_width;
    double scale_y = static_cast<double>(out_height) / in_height;
    double src_x = (x + 0.5) / scale_x - 0.5;
    double src_y = (y + 0.5) / scale_y - 0.5;

    // Compute the window boundaries
    int x_start = static_cast<int>(floor(src_x - a + 1));
    int x_end = static_cast<int>(floor(src_x + a));
    int y_start = static_cast<int>(floor(src_y - a + 1));
    int y_end = static_cast<int>(floor(src_y + a));

    x_start = max(0, x_start);
    x_end = min(in_width - 1, x_end);
    y_start = max(0, y_start);
    y_end = min(in_height - 1, y_end);

    for (int j = y_start; j <= y_end; ++j) {
        double dist_y = src_y - j;
        double wy = lanczos_kernel(dist_y, a);
        for (int i = x_start; i <= x_end; ++i) {
            double dist_x = src_x - i;
            double wx = lanczos_kernel(dist_x, a);
            double weight = wx * wy;

            int idx_in = (j * in_width + i) * channels;

            for (int c = 0; c < channels; ++c) {
                sum[c] += weight * input[idx_in + c];
            }
            sum_weight += weight;
        }
    }

    if (sum_weight > 0.0) {
        for (int c = 0; c < channels; ++c) {
            sum[c] /= sum_weight;
            sum[c] = fmin(fmax(sum[c], 0.0), 255.0);
            output[idx_out + c] = static_cast<unsigned char>(sum[c] + 0.5);
        }
    }
    else {
        for (int c = 0; c < channels; ++c) {
            output[idx_out + c] = 0;
        }
    }
}



// Structure to hold device copies of AxisParam data
//struct DeviceAxisParam {
//    int* start;
//    int* length;
//    double* weight;
//    int* index;
//
//    explicit DeviceAxisParam(const AxisParam& param) {
//        hipMalloc(&start, sizeof(int) * param.start.size());
//        hipMalloc(&length, sizeof(int) * param.length.size());
//        hipMalloc(&weight, sizeof(double) * param.weight.size());
//        hipMalloc(&index, sizeof(int) * param.index.size());
//
//        hipMemcpy(start, param.start.data(), sizeof(int) * param.start.size(), hipMemcpyHostToDevice);
//        hipMemcpy(length, param.length.data(), sizeof(int) * param.length.size(), hipMemcpyHostToDevice);
//        hipMemcpy(weight, param.weight.data(), sizeof(double) * param.weight.size(), hipMemcpyHostToDevice);
//        hipMemcpy(index, param.index.data(), sizeof(int) * param.index.size(), hipMemcpyHostToDevice);
//    }
//
//    ~DeviceAxisParam() {
//        hipFree(start);
//        hipFree(length);
//        hipFree(weight);
//        hipFree(index);
//    }
//};

// Function to perform Lanczos resampling on the GPU
// Function to perform Lanczos resampling on the GPU
void lanczos_resample_cuda(
    const unsigned char* h_input,
    unsigned char* h_output,
    int in_width,
    int in_height,
    int out_width,
    int out_height,
    int channels,
    int a)
{
    // Allocate device memory for input and output images
    unsigned char* d_input = nullptr;
    unsigned char* d_output = nullptr;

    size_t input_size = in_width * in_height * channels * sizeof(unsigned char);
    size_t output_size = out_width * out_height * channels * sizeof(unsigned char);

    hipError_t err;

    err = hipMalloc(&d_input, input_size);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed for input: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_output, output_size);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed for output: " << hipGetErrorString(err) << std::endl;
        hipFree(d_input);
        return;
    }

    // Copy input image to device
    err = hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy failed for input: " << hipGetErrorString(err) << std::endl;
        hipFree(d_input);
        hipFree(d_output);
        return;
    }

    // Launch the kernel
    dim3 blockSize(16, 16);
    dim3 gridSize((out_width + blockSize.x - 1) / blockSize.x,
                  (out_height + blockSize.y - 1) / blockSize.y);

    lanczos_resample_kernel<<<gridSize, blockSize>>>(
        d_input,
        d_output,
        in_width,
        in_height,
        out_width,
        out_height,
        channels,
        a);

    // Check for kernel errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_input);
        hipFree(d_output);
        return;
    }

    // Wait for GPU to finish
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA device synchronize error: " << hipGetErrorString(err) << std::endl;
        hipFree(d_input);
        hipFree(d_output);
        return;
    }

    // Copy output image back to host
    err = hipMemcpy(h_output, d_output, output_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy failed for output: " << hipGetErrorString(err) << std::endl;
        hipFree(d_input);
        hipFree(d_output);
        return;
    }

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}

